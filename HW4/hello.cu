
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel () {
    int ID = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %d\n", ID);
}

int main() {
    hello_kernel<<<2, 4>>>();
    hipDeviceSynchronize ();
    return 0;
}
