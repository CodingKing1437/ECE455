
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel () {
    int ID = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %d", ID);
}

int main() {
    hello_kernel<<<2, 4>>>();
    hipDeviceSynchronize ();
    return 0;
}